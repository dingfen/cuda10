#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define CUDA_ERROR_WRAP(x) { gpuAssert((x), __FILE__, __LINE__); }
#define N (1 * 1024)

inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    hipDeviceProp_t prop;
    int count = 0;

    CUDA_ERROR_WRAP(hipGetDeviceCount(&count))

    for (int i = 0; i < count; i++) {
        CUDA_ERROR_WRAP(hipGetDeviceProperties(&prop, i))
        printf("  --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Device async Engine Count: %d\n", prop.asyncEngineCount);
        printf("Kernel execution timeout:  ");
        if (prop.kernelExecTimeoutEnabled)
            printf("Enabled.\n");
        else
            printf("Disabled.\n");
        printf("Device is ");
        if (prop.integrated)
            printf("integrated.\n");
        else
            printf("discreted.\n");
        printf("  --- Memory Information for device %d ---\n", i);
        printf("Total global mem: %lu\n", prop.totalGlobalMem);
        printf("Total constant mem: %lu\n", prop.totalConstMem);
        printf("Max mem pitch:  %ld\n", prop.memPitch);
        printf("memory Clock Rate: %d\n", prop.memoryClockRate);
        printf("memory Bus Width: %d\n", prop.memoryBusWidth);
        printf("Texture Alignment:  %ld\n", prop.textureAlignment);

        printf("  --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
        printf("concurrent execute kernels count: %d\n", prop.concurrentKernels);
        printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp:  %ld\n", prop.sharedMemPerBlock);
        printf("Threads in wrap:  %d\n", prop.warpSize);
        printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
        printf("Max threads per MP: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("Max size of shared mem per MP: %lu\n", prop.sharedMemPerMultiprocessor);
        printf("Max threads dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
}
