#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define CUDA_ERROR_WRAP(x) gpuAssert((x), __FILE__, __LINE__)
const int N = 33*1024;

inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void addKernel(int* c, int* a, int* b)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void add_fp32(float* c, float* a, float* b)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

extern "C" void addWithCUDA(int* c, int* a, int* b) {
    addKernel<<<128, 128 >>>(c, a, b);
}

extern "C" void add_fp32_CUDA(float* c, float* a, float* b) {
    add_fp32<<<128, 128 >>>(c, a, b);
}

int main()
{
    float a[N];
    float b[N];
    float c[N];
    float* dev_a, * dev_b, * dev_c;
    bool success = true;

    CUDA_ERROR_WRAP(hipMalloc(&dev_a, N * sizeof(float)));
    CUDA_ERROR_WRAP(hipMalloc(&dev_b, N * sizeof(float)));
    CUDA_ERROR_WRAP(hipMalloc(&dev_c, N * sizeof(float)));

    srand(time(0));
    for (int i = 0; i < N; i++) {
        a[i] = rand() * (4.0 / RAND_MAX) - 2.0;
        b[i] = rand() * (4.0 / RAND_MAX) - 2.0;
    }

    CUDA_ERROR_WRAP(hipSetDevice(0));

    CUDA_ERROR_WRAP(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_ERROR_WRAP(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add_fp32_CUDA(dev_c, dev_a, dev_b);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    CUDA_ERROR_WRAP(hipDeviceSynchronize());

    CUDA_ERROR_WRAP(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < N; i++) {
        if (a[i] + b[i] != c[i]) {
            printf("Error: %f + %f != %f at index %d\n", a[i], b[i], c[i], i);
            success = false;
        }
    }

    if (success) {
        printf("We did it!\n");
    }

    CUDA_ERROR_WRAP(hipDeviceReset());
   
Error:
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
