
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define GPUAssert(x) gpuAssert((x), __FILE__, __LINE__)

inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

int main()
{
    float time;
    hipEvent_t start, stop;
    GPUAssert(hipEventCreate(&start));
    GPUAssert(hipEventCreate(&stop));
    GPUAssert(hipEventRecord(start, 0));

    GPUAssert(hipEventRecord(stop, 0));
    GPUAssert(hipEventElapsedTime(&time, start, stop));
}

