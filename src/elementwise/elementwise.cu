#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"

#include <stdio.h>
#include <stdlib.h>

#define GPUAssert(x) gpuAssert((x), __FILE__, __LINE__)

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

__global__ void relu_kernel(float *input, float *output) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    output[idx] = input[idx] < 0 ? 0 : input[idx];
}

int main() {
    float *input;
    float *output;
    int32_t elem_cnt = 3 * 224 * 224;
    int loop_times = 100;
    hipEvent_t startEvent, stopEvent;
    float ms; // elapsed time in milliseconds

    hipMalloc(&input, sizeof(float) * elem_cnt);
    hipMalloc(&output, sizeof(float) * elem_cnt);
    GPUAssert(hipEventCreate(&startEvent));
    GPUAssert(hipEventCreate(&stopEvent));

    int32_t thread_num = 256;
    int32_t grid_size = (elem_cnt + thread_num - 1) / thread_num;
    
    hipProfilerStart();
    GPUAssert(hipEventRecord(startEvent, 0));
    for(int i = 0; i < loop_times; i++) {
        relu_kernel<<<grid_size, thread_num>>>(input, output);
        GPUAssert(hipStreamSynchronize(0));
    }
    GPUAssert(hipEventRecord(stopEvent, 0));
    GPUAssert(hipEventSynchronize(stopEvent));
    hipProfilerStop();
    GPUAssert(hipEventElapsedTime(&ms, startEvent, stopEvent));
    printf("Time for sequential transfer and execute (ms): %f\n", ms);

    hipFree(input);
    hipFree(output);
    return 0;
}