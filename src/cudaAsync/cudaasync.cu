
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define GPUAssert(x) gpuAssert((x), __FILE__, __LINE__)

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline void gpuAssert(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

__global__ void kernel(float *a, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  float x = (float)i;
  float s = sinf(x);
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s * s + c * c);
}

float maxError(float *a, int n)
{
  float maxE = 0;
  for (int i = 0; i < n; i++)
  {
    float error = fabs(a[i] - 1.0f);
    if (error > maxE)
      maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv)
{
  const int blockSize = 256, nStreams = 4;
  const int n = 4 * 1024 * blockSize * nStreams;
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);

  int devId = 0;
  if (argc > 1)
    devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  GPUAssert(hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  GPUAssert(hipSetDevice(devId));

  // allocate pinned host memory and device memory
  float *a, *d_a;
  GPUAssert(hipHostMalloc((void **)&a, bytes, hipHostMallocDefault)); // host pinned
  GPUAssert(hipMalloc((void **)&d_a, bytes));   // device

  float ms; // elapsed time in milliseconds

  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  GPUAssert(hipEventCreate(&startEvent));
  GPUAssert(hipEventCreate(&stopEvent));
  GPUAssert(hipEventCreate(&dummyEvent));
  for (int i = 0; i < nStreams; ++i)
    GPUAssert(hipStreamCreate(&stream[i]));

  // baseline case - sequential transfer and execute
  memset(a, 0, bytes);
  GPUAssert(hipEventRecord(startEvent, 0));
  GPUAssert(hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice));
  kernel<<<n / blockSize, blockSize>>>(d_a, 0);
  GPUAssert(hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost));
  GPUAssert(hipEventRecord(stopEvent, 0));
  GPUAssert(hipEventSynchronize(stopEvent));
  GPUAssert(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("Time for sequential transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 1: loop over {copy, kernel, copy}
  memset(a, 0, bytes);
  GPUAssert(hipEventRecord(startEvent, 0));
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    GPUAssert(hipMemcpyAsync(&d_a[offset], &a[offset],
                              streamBytes, hipMemcpyHostToDevice,
                              stream[i]));
    kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    GPUAssert(hipMemcpyAsync(&a[offset], &d_a[offset],
                              streamBytes, hipMemcpyDeviceToHost,
                              stream[i]));
  }
  GPUAssert(hipEventRecord(stopEvent, 0));
  GPUAssert(hipEventSynchronize(stopEvent));
  GPUAssert(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // asynchronous version 2:
  // loop over copy, loop over kernel, loop over copy
  memset(a, 0, bytes);
  GPUAssert(hipEventRecord(startEvent, 0));
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    GPUAssert(hipMemcpyAsync(&d_a[offset], &a[offset],
                              streamBytes, hipMemcpyHostToDevice,
                              stream[i]));
  }
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
  }
  for (int i = 0; i < nStreams; ++i)
  {
    int offset = i * streamSize;
    GPUAssert(hipMemcpyAsync(&a[offset], &d_a[offset],
                              streamBytes, hipMemcpyDeviceToHost,
                              stream[i]));
  }
  GPUAssert(hipEventRecord(stopEvent, 0));
  GPUAssert(hipEventSynchronize(stopEvent));
  GPUAssert(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("Time for asynchronous V2 transfer and execute (ms): %f\n", ms);
  printf("  max error: %e\n", maxError(a, n));

  // cleanup
  GPUAssert(hipEventDestroy(startEvent));
  GPUAssert(hipEventDestroy(stopEvent));
  GPUAssert(hipEventDestroy(dummyEvent));
  for (int i = 0; i < nStreams; ++i)
    GPUAssert(hipStreamDestroy(stream[i]));
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}