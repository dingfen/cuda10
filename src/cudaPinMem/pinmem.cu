
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define GPUAssert(x) gpuAssert((x), __FILE__, __LINE__)

inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}


void profileCopies(float *h_a, float *h_b, float *d, 
                   unsigned int  n, const char *desc) {
    printf("\n%s transfers\n", desc);
    unsigned int bytes = n * sizeof(float);

    // events for timing
    hipEvent_t startEvent, stopEvent;

    GPUAssert( hipEventCreate(&startEvent) );
    GPUAssert( hipEventCreate(&stopEvent) );

    GPUAssert( hipEventRecord(startEvent, 0) );
    GPUAssert( hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice) );
    GPUAssert( hipEventRecord(stopEvent, 0) );
    GPUAssert( hipEventSynchronize(stopEvent) );

    float time;
    GPUAssert( hipEventElapsedTime(&time, startEvent, stopEvent) );
    printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

    GPUAssert( hipEventRecord(startEvent, 0) );
    GPUAssert( hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost) );
    GPUAssert( hipEventRecord(stopEvent, 0) );
    GPUAssert( hipEventSynchronize(stopEvent) );

    GPUAssert( hipEventElapsedTime(&time, startEvent, stopEvent) );
    printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

    for (int i = 0; i < n; ++i) {
      if (h_a[i] != h_b[i]) {
        printf("*** %s transfers failed ***\n", desc);
        break;
      }
    }

    // clean up events
    GPUAssert( hipEventDestroy(startEvent) );
    GPUAssert( hipEventDestroy(stopEvent) );
}


int main() {
    unsigned int nElements = 4*1024*1024;
    const unsigned int bytes = nElements * sizeof(float);
    
    // host arrays
    float *h_aPageable, *h_bPageable;   
    float *h_aPinned, *h_bPinned;
    
    // device array
    float *d_a;
    
    // allocate and initialize
    h_aPageable = (float*)malloc(bytes);                    // host pageable
    h_bPageable = (float*)malloc(bytes);                    // host pageable
    GPUAssert( hipHostMalloc((void**)&h_aPinned, bytes, hipHostMallocDefault) ); // host pinned
    GPUAssert( hipHostMalloc((void**)&h_bPinned, bytes, hipHostMallocDefault) ); // host pinned
    GPUAssert( hipMalloc((void**)&d_a, bytes) );           // device
    
    for (int i = 0; i < nElements; ++i) h_aPageable[i] = i;      
    memcpy(h_aPinned, h_aPageable, bytes);
    memset(h_bPageable, 0, bytes);
    memset(h_bPinned, 0, bytes);
    
    // output device info and transfer size
    hipDeviceProp_t prop;
    GPUAssert( hipGetDeviceProperties(&prop, 0) );
    
    printf("\nDevice: %s\n", prop.name);
    printf("Transfer size (MB): %d\n", bytes / (1024 * 1024));
    
    // perform copies and report bandwidth
    profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
    profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");
    
    printf("n");
    
    // cleanup
    hipFree(d_a);
    hipHostFree(h_aPinned);
    hipHostFree(h_bPinned);
    free(h_aPageable);
    free(h_bPageable);
    
    return 0;
}